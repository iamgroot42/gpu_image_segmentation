#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <IL/il.h>
#include <IL/ilu.h>

#define BLOCK_SIZE 32
#define HYPERm 2
#define HYPERk 4
#define LAMBDA 2

using namespace std;

__device__ unsigned long long B_function(int x, int y){
	return (x - y) * (x - y);
}

__device__  unsigned long long R_function(int x, int y){
	//Object 
	if(y == 1){
		return 1;
	}
	return 2;
}

struct Pixel{
	int pixel_value, hard_constraint, height;
	unsigned long long neighbor_capacities[10]; //Stored in row major form, followed by source and sink
	unsigned long long int excess;
	bool is_active;
	Pixel(){
		this -> hard_constraint = 0;
		this -> height = 0;
		this -> excess = 0;
		this -> is_active = false;
	}
};

struct Terminal{
	unsigned long long int excess;
	bool is_active;
	int height;
	Terminal(){
		this -> is_active = false;
		this -> height = 0;
		this -> excess = 0;
	}
};

void saveImage(const char* filename, int width, int height, unsigned char * bitmap){
	ILuint imageID = ilGenImage();
	ilBindImage(imageID);
	ilTexImage(width, height, 0, 1, IL_LUMINANCE, IL_UNSIGNED_BYTE, bitmap);
	iluFlipImage();
	ilEnable(IL_FILE_OVERWRITE);
	ilSave(IL_PNG, filename);
	fprintf(stderr, "Image saved as: %s\n", filename);
}

ILuint loadImage(const char *filename, unsigned char ** bitmap, int &width, int &height){
	ILuint imageID = ilGenImage();
	ilBindImage(imageID);
	ILboolean success = ilLoadImage(filename);
	if (!success) return 0;

	width = ilGetInteger(IL_IMAGE_WIDTH);
	height = ilGetInteger(IL_IMAGE_HEIGHT);
	printf("Width: %d\t Height: %d\n", width, height);
	*bitmap = ilGetData();
	return imageID;
}

__global__ void push(Pixel *image_graph, unsigned long long *F, int height, int width, int *convergence_flag){
	int i = (threadIdx.x + blockIdx.x * blockDim.x) + 1;
	int j = (threadIdx.y + blockDim.y * blockIdx.y) + 1;

	if (i <= height && j <= width)
	{
		int locali = (i - 1) % BLOCK_SIZE, localj = (j - 1) % BLOCK_SIZE;

		__shared__ int shared_heights[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
		__shared__ int shared_excess[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
		__shared__ int block_flag ;
		block_flag = 0;
		
		shared_heights[locali + 1][localj + 1] = image_graph[i * width + j].height;
		shared_excess[locali + 1][localj + 1] = image_graph[i * width + j].excess;

		//Boundary pixels of grid
		if(locali == 0){
			shared_excess[0][localj + 1] = image_graph[(i - 1) * width + j].excess;
			shared_heights[0][localj + 1] = image_graph[(i - 1) * width + j].height;
			if(localj == 0){
				shared_excess[0][0] = image_graph[(i - 1) * width + (j - 1)].excess;
				shared_heights[0][BLOCK_SIZE + 1] = image_graph[(i - 1) * width + (j + 1)].height;
			}
		}
		else if(locali == BLOCK_SIZE - 1){
			shared_excess[BLOCK_SIZE + 1][localj + 1] = image_graph[(i + 1) * width + j].excess;
			shared_heights[BLOCK_SIZE + 1][localj + 1] = image_graph[(i + 1) * width + j].height;
			if(localj == 0){
				shared_excess[BLOCK_SIZE + 1][0] = image_graph[(i + 1) * width + (j - 1)].excess;
				shared_heights[BLOCK_SIZE + 1][BLOCK_SIZE + 1] = image_graph[(i + 1) * width + (j + 1)].height;
			}
		}
		else if(localj == 0){
			shared_excess[locali + 1][0] = image_graph[i * width + (j - 1)].excess;
			shared_heights[locali + 1][0] = image_graph[i * width + (j - 1)].height;
		}
		else if(localj == BLOCK_SIZE - 1){
			shared_excess[locali + 1][BLOCK_SIZE + 1] = image_graph[i * width + (j + 1)].excess;
			shared_heights[locali + 1][BLOCK_SIZE + 1] = image_graph[i * width + (j + 1)].height;
		}
		__syncthreads();

		// Row major traversal of neighbors of a pixel (i,j)
		int x_offsets[] = {-1, -1, -1, 0, 0, 1, 1, 1};
		int y_offsets[] = {-1, 0, 1, -1, 1, -1, 0, 1};

		int thread_flag = 0;
		int dest_x, dest_y;

		// Check spatial neighbors
		for(int l = 0; l < 8; l++){
			dest_x = (locali + 1) + x_offsets[l];
			dest_y = (localj + 1) + y_offsets[l];
			if(shared_heights[dest_x][dest_y] + 1 == shared_heights[locali + 1][localj + 1]){
				shared_excess[dest_x][dest_y] += shared_excess[locali + 1][localj + 1]; //push e(u) to eligible neighbors
				thread_flag = 1;
			}
		}

		// TODO: Run same condition as above for source, sink

		__syncthreads();
		//store excess flow in a global 'F' array
		F[i * width + j] = shared_excess[locali + 1][localj + 1];

		// Update flags
		atomicOr(&block_flag, thread_flag);
		__syncthreads();

		if(threadIdx.x == 0 && threadIdx.y == 0){
			atomicOr(convergence_flag, block_flag);
		}
		printf("%d ", *convergence_flag);
	}
}


__global__ void pull(Pixel *image_graph, unsigned long long *F, int height, int width){
	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int j = threadIdx.y + blockDim.y * blockIdx.y + 1;

	// Should be <=, but fails for that
	if (i < height && j < width)
	{
		unsigned long long aggregate_flow = 0;
		// Row major traversal of neighbors of a pixel (i,j)
		int x_offsets[] = {-1, -1, -1, 0, 0, 1, 1, 1};
		int y_offsets[] = {-1, 0, 1, -1, 1, -1, 0, 1};

		int dest_x, dest_y;

		// Check spatial neighbors
		for(int k = 0; k < 8; k++){
			dest_x = i + x_offsets[k];
			dest_y = j + y_offsets[k];
			aggregate_flow += F[dest_x * width + dest_y];
		}

		//TODO: Run same condition as above for source, sink

		image_graph[i * width + j].excess += aggregate_flow;
	}
}

__global__ void localRelabel(Pixel *image_graph, int height, int width){
	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int j = threadIdx.y + blockDim.y * blockIdx.y + 1;
	int locali = (i - 1) % BLOCK_SIZE, localj = (j - 1) % BLOCK_SIZE;

	if (i <= height && j <= width){
		__shared__ int shared_heights[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
		// __shared__ bool shared_flags[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

		shared_heights[locali + 1][localj + 1] = image_graph[i * width + j].height;
		// shared_flags[locali + 1][localj + 1] = image_graph[i * width + j].is_active;

		//Boundary pixels of grid
		if(locali == 0){
				shared_heights[0][localj + 1] = image_graph[(i - 1) * width + j].height;
			if(localj == 0){
				shared_heights[0][BLOCK_SIZE + 1] = image_graph[(i - 1) * width + (j + 1)].height;
			}
		}
		else if(locali == BLOCK_SIZE - 1){
				shared_heights[BLOCK_SIZE + 1][localj + 1] = image_graph[(i + 1) * width + j].height;
			if(localj == 0){
				shared_heights[BLOCK_SIZE + 1][BLOCK_SIZE + 1] = image_graph[(i + 1) * width + (j + 1)].height;
			}	
		}
		else if(localj == 0){
			shared_heights[locali + 1][0] = image_graph[i * width + (j - 1)].height;
		}
		else if(localj == BLOCK_SIZE - 1){
			shared_heights[locali + 1][BLOCK_SIZE + 1] = image_graph[i * width + (j + 1)].height;
		}

		__syncthreads();

		// Row major traversal of neighbors of a pixel (i,j)
		int x_offsets[] = {-1, -1, -1, 0, 0, 1, 1, 1};
		int y_offsets[] = {-1, 0, 1, -1, 1, -1, 0, 1};

		int dest_x, dest_y;
		int min_height = INT_MAX;

		// Check spatial neighbors
		for(int l = 0; l < 8; l++){
			dest_x = (locali + 1) + x_offsets[l];
			dest_y = (localj + 1) + y_offsets[l];
			if( image_graph[dest_x * width + dest_y].is_active){
				min_height = min(min_height, shared_heights[dest_x][dest_y]);
			}
		}

		// Run same condition as above for source, sink

		image_graph[i * width + j].height = min_height + 1;
	}
}

__global__ void globalRelabel(Pixel *image_graph, int height, int width, int iteration){
	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int j = threadIdx.y + blockDim.y * blockIdx.y + 1;

	if (i <= height && j <= width){

		int locali = (i - 1) % BLOCK_SIZE, localj = (j - 1) % BLOCK_SIZE;

		//No divergence
		if(iteration == 1){
			for (int l = 0; l < 8; l++)
				if(image_graph[i * width + j].neighbor_capacities[l] > image_graph[i * width + j].excess){
					image_graph[i * width + j].height = 1;
			}
		}
		else{
			__shared__ int shared_heights[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
			
			shared_heights[locali + 1][localj + 1] = image_graph[i * width + j].height;

			//Boundary pixels of grid
			if(locali == 0){
				shared_heights[0][localj + 1] = image_graph[(i - 1) * width + j].height;
				if(localj == 0){
					shared_heights[0][BLOCK_SIZE + 1] = image_graph[(i - 1) * width + (j + 1)].height;
				}
			}
			else if(locali == BLOCK_SIZE - 1){
				shared_heights[BLOCK_SIZE + 1][localj + 1] = image_graph[(i + 1) * width + j].height;
				if(localj == 0){
					shared_heights[BLOCK_SIZE + 1][BLOCK_SIZE + 1] = image_graph[(i + 1) * width + (j + 1)].height;
				}	
			}
			else if(localj == 0){
				shared_heights[locali + 1][0] = image_graph[i * width + (j - 1)].height;
			}
			else if(localj == BLOCK_SIZE - 1){
				shared_heights[locali + 1][BLOCK_SIZE + 1] = image_graph[i * width + (j + 1)].height;
			}

			__syncthreads();

			bool satisfied = false;
			int dest_x, dest_y;

			// Row major traversal of neighbors of a pixel (i,j)
			int x_offsets[] = {-1, -1, -1, 0, 0, 1, 1, 1};
			int y_offsets[] = {-1, 0, 1, -1, 1, -1, 0, 1};

			for(int i1 = 0; i1 < 8; i1++){
				dest_x = (locali + 1) + x_offsets[i1];
				dest_y = (localj + 1) + y_offsets[i1];
				if(shared_heights[dest_x][dest_y] == iteration){
					satisfied = true;
					break;
				}
			}

			if(satisfied){
				shared_heights[locali + 1][localj + 1] = iteration + 1;
				image_graph[i * width + j].height = iteration + 1;
			}
		}
	}
}

__global__ void initNeighbors(Pixel *imagegraph, unsigned char* raw_image, int height, int width, unsigned long long int* K){

	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int j = threadIdx.y + blockDim.y * blockIdx.y + 1;

	int locali = (i - 1) % BLOCK_SIZE, localj = (j - 1) % BLOCK_SIZE;

	if (i <= height && j <= width)
	{
		__shared__ unsigned long long block_pixels[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
		imagegraph[i * width + j].pixel_value = raw_image[(i - 1) * width + j - 1];
		
		// 1-indexing for block_pixels
		block_pixels[locali + 1][localj + 1] = imagegraph[i * width + j].pixel_value;

		//Boundary pixels of grid
		if(locali == 0)
		{
			block_pixels[locali][localj + 1] = imagegraph[(i - 1) * width + j].pixel_value;
			if(localj == 0)
				block_pixels[locali][localj] = imagegraph[(i - 1) * width + (j - 1)].pixel_value;
		}
		else if(locali == BLOCK_SIZE - 1)
		{
			assert((i + 1) * width + j + 1 < height * width);
			block_pixels[BLOCK_SIZE + 1][localj + 1] = imagegraph[(i + 1) * width + j].pixel_value;
			if(localj == BLOCK_SIZE - 1)
				block_pixels[BLOCK_SIZE + 1][BLOCK_SIZE + 1] = imagegraph[(i + 1) * width + (j + 1)].pixel_value;
		}
		else if(localj == 0){
			block_pixels[locali + 1][0] = imagegraph[i * width + (j - 1)].pixel_value;
		}
		else if(localj == BLOCK_SIZE - 1){
			block_pixels[locali + 1][BLOCK_SIZE + 1] = imagegraph[i * width + (j + 1)].pixel_value;
		}
		__syncthreads();

		// Row major traversal of neighbors of a pixel (i,j)
		int x_offsets[] = {-1, -1, -1, 0, 0, 1, 1, 1};
		int y_offsets[] = {-1, 0, 1, -1, 1, -1, 0, 1};

		unsigned long long int max_k = 0;
		unsigned long long edge_weight = 0;
		int dest_x, dest_y;

		for(int k = 0; k < 8; k++){
			dest_x = (locali + 1) + x_offsets[k];
			dest_y = (localj + 1) + y_offsets[k];
			edge_weight = B_function(block_pixels[locali + 1][localj + 1], block_pixels[dest_x][dest_y]);
			imagegraph[i * width + j].neighbor_capacities[k] = edge_weight;
			max_k += edge_weight;
		}
		max_k++;

		__shared__ unsigned long long int blockmax;

		if(threadIdx.x == 0 && threadIdx.y == 0){
			blockmax = INT_MAX;
		}
		__syncthreads();
		atomicMax(&blockmax, max_k);
		__syncthreads();

		if(threadIdx.x == 0 && threadIdx.y == 0){
			atomicMax(K, blockmax);
		}
	}
}

//Also accept hard and soft constraints array
__global__ void initConstraints(Pixel *imagegraph, int height, int width, unsigned long long K){

	int i = threadIdx.x + blockIdx.x * blockDim.x + 1;
	int j = threadIdx.y + blockDim.y * blockIdx.y + 1;

	if (i <= height && j <= height)
	{
		// {p,S} edge
		imagegraph[i * width + j].neighbor_capacities[8] = (imagegraph[i * width + j].hard_constraint == 0) * K
										+ (imagegraph[i * width + j].hard_constraint == 1) * LAMBDA * R_function(imagegraph[i * width + j].pixel_value, -1);

		// {p,T} edge
		imagegraph[i * width + j].neighbor_capacities[9] = (imagegraph[i * width + j].hard_constraint == -1) * K
										+ (imagegraph[i * width + j].hard_constraint == 0) * LAMBDA * R_function(imagegraph[i * width + j].pixel_value, 1);
	}
}


int main(int argc, char* argv[]){
	int width, height;
	unsigned long long* K = new unsigned long long;
	*K = LLONG_MAX;
	int* convergence_flag = new int, *convergence_flag_gpu;
	*convergence_flag = 0;

	unsigned char *image, *cuda_image;
	unsigned long long *K_gpu, *F_gpu;
	Pixel *image_graph, *cuda_image_graph;
	Terminal *source, *sink, *cuda_source, *cuda_sink;
	
	ilInit();

	ILuint image_id = loadImage(argv[1], &image, width, height);
	int pixel_memsize = (width + 1) * (height + 1) * sizeof(Pixel);
	if(image_id == 0) {fprintf(stderr, "Error while reading image... aborting.\n"); exit(0);}

	//Pixel graph with padding to avoid convergence in kernels for boundary pixels
	image_graph = (Pixel*)malloc(pixel_memsize);
	source = new Terminal;
	sink = new Terminal;	

	hipMalloc((void**)&F_gpu, (width + 1) * (height + 1) * sizeof(unsigned long long));
	hipMalloc((void**)&convergence_flag_gpu, sizeof(int));
	hipMalloc((void**)&cuda_image_graph, pixel_memsize);
	hipMalloc((void**)&cuda_image, width * height * sizeof(unsigned char));
	hipMalloc((void**)&cuda_image_graph, pixel_memsize);
	hipMalloc((void**)&K_gpu, sizeof(unsigned long long));
	hipMalloc((void**)&cuda_source, sizeof(Terminal));
	hipMalloc((void**)&cuda_sink, sizeof(Terminal));
	
	//Set properties of source and sink nodes

	hipMemcpy(cuda_image_graph, image_graph, pixel_memsize, hipMemcpyHostToDevice);
	hipMemcpy(cuda_image, image, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(K_gpu, K, sizeof(unsigned long long), hipMemcpyHostToDevice);
	hipMemcpy(convergence_flag_gpu, convergence_flag, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_source, source, sizeof(Terminal), hipMemcpyHostToDevice);
        hipMemcpy(cuda_sink, sink, sizeof(Terminal), hipMemcpyHostToDevice);


	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks(height / BLOCK_SIZE + 1, width / BLOCK_SIZE + 1);

	// Load weights in graph using kernel call/host loops
	initNeighbors<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, cuda_image, height, width, K_gpu);
	assert(hipSuccess == hipGetLastError());
	printf("Initialized spatial weight values\n");
	initConstraints<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, height, width, *K);
	assert(hipSuccess == hipGetLastError());
	printf("Initialized terminal weight values\n");

	int iteration = 1;
	while((*convergence_flag) || (!(*convergence_flag && iteration == 1))){
		for(int i = 0; i < HYPERk; i++){
			for(int j = 0; j < HYPERm; j++){
				push<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, F_gpu, height, width, convergence_flag_gpu);
				assert(hipSuccess == hipGetLastError());
				printf("Local push operation\n");
				pull<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, F_gpu, height, width);
				assert(hipSuccess == hipGetLastError());
				printf("Local pull operation\n");
				hipMemcpy(convergence_flag, convergence_flag_gpu, sizeof(int), hipMemcpyDeviceToHost);
				printf("%d\n", *convergence_flag);
			}
			localRelabel<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, height, width);
			assert(hipSuccess == hipGetLastError());
			printf("Local relabel operation\n");
		}
		globalRelabel<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, height, width, iteration);
		assert(hipSuccess == hipGetLastError());
		printf("Global relabel operation\n");
		iteration++;
		printf("Completed iteration %d\n\n", iteration);
	}

	printf("Done with algorithm\n");
	// Load segmented image from graph using another kernel and display it

	return 0;
}
