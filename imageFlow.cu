#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <IL/il.h>
#include <IL/ilu.h>

#define BLOCK_SIZE 64
#define HYPERm 2
#define HYPERk 4
#define LAMBDA 2


using namespace std;

// Global temporary storage F
__device__ unsigned long long** F;

unsigned long long B_function(int x, int y){
	return (x - y) * (x - y);
}

unsigned long long R_function(int x, int y){
	//Object 
	if(y == 1){
		return 1;
	}
	return 2;
}

struct Pixel
{
	int pixel_value, hard_constraint, height;
	unsigned long long neighbor_capacities[10]; //Stored in row major form, followed by source and sink
	unsigned long long int  excess;
	bool is_active;
	Pixel()
	{
		this -> hard_constraint = 0;
		this -> height = 0;
		this -> excess = 0;
		this -> is_active = false;
	}
};

void saveImage(const char* filename, int width, int height, unsigned char * bitmap)
{
	ILuint imageID = ilGenImage();
	ilBindImage(imageID);
	ilTexImage(width, height, 0, 1,IL_LUMINANCE, IL_UNSIGNED_BYTE, bitmap);
	iluFlipImage();
	ilEnable(IL_FILE_OVERWRITE);
	ilSave(IL_PNG, filename);
	fprintf(stderr, "Image saved as: %s\n", filename);
}

ILuint loadImage(const char *filename, unsigned char ** bitmap, int &width, int &height)
{
	ILuint imageID = ilGenImage();
	ilBindImage(imageID);
	ILboolean success = ilLoadImage(filename);
	if (!success) return 0;

	width = ilGetInteger(IL_IMAGE_WIDTH);
	height = ilGetInteger(IL_IMAGE_HEIGHT);
	printf("Width: %d\t Height: %d\n", width, height);
	*bitmap = ilGetData();
	return imageID;
}

__global__ void push(Pixel *image_graph, int height, int width)
{
	int offset = height + width;
	int i = (threadIdx.x + blockIdx.x * blockDim.x) + offset;
	int j = (threadIdx.y + blockDim.y * blockIdx.y) + offset;

	int locali = i%BLOCK_SIZE, localj =j%BLOCK_SIZE;

	__shared__ int shared_heights[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
	__shared__ int shared_excess[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

	shared_heights[locali + 1][localj + 1] = image_graph[i * width + j].height;
	shared_excess[locali + 1][localj + 1] = image_graph[i * width + j].excess;

	if(locali == 0){
		shared_excess[0][localj + 1] = image_graph[(i - 1) * width + j].excess;
		shared_heights[0][localj + 1] = image_graph[(i - 1) * width + j].height;
		if(localj == 0){
			shared_excess[0][0] = image_graph[(i - 1) * width + j - 1].excess;
			shared_heights[0][BLOCK_SIZE + 1] = image_graph[(i - 1) * width + j + 1].height;
		}
	}
	else if(localj == BLOCK_SIZE - 1){
		shared_excess[BLOCK_SIZE + 1][localj + 1] = image_graph[(i + 1) * width + j].excess;
		shared_heights[BLOCK_SIZE + 1][localj + 1] = image_graph[(i + 1) * width + j].height;
		if(localj == 0){
			shared_excess[BLOCK_SIZE + 1][0] = image_graph[(i + 1) * width + j - 1].excess;
			shared_heights[BLOCK_SIZE + 1][BLOCK_SIZE + 1] = image_graph[(i + 1) * width + j + 1].height;
		}	
	}
	else if(localj == 0){
		shared_excess[locali + 1][0] = image_graph[i * width + j - 1].excess;
		shared_heights[locali + 1][0] = image_graph[i * width + j - 1].height;
	}
	else if(localj == BLOCK_SIZE - 1){
		shared_excess[locali + 1][BLOCK_SIZE + 1] = image_graph[i * width + j + 1].excess;
		shared_heights[locali + 1][BLOCK_SIZE + 1] = image_graph[i * width + j + 1].height;
	}
	__syncthreads();

	// Row major traversal of neighbors of a pixel (i,j)
	int x_offsets[] = {-1,-1,-1,0,0,1,1,1};
	int y_offsets[] = {-1,0,1,-1,1,-1,0,1};

	int dest_x, dest_y;
	// Check spatial neighbors
	for(int i=0;i<8;i++){
		dest_x = locali + x_offsets[i] + 1;
		dest_y = localj + y_offsets[i] + 1;
		if(shared_heights[dest_x][dest_y] + 1 == shared_heights[localj][localj]){
			shared_excess[dest_x][dest_y] += shared_excess[locali][localj]; //push e(u) to eligible neighbors
		}
	}
	// Run same condition as above for source, sink

	__syncthreads();
	//store excess flow in a global 'F' array
}


__global__ void pull(Pixel *image_graph, int height, int width)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;

	// Row major traversal of neighbors of a pixel (i,j)
	int x_offsets[] = {-1,-1,-1,0,0,1,1,1};
	int y_offsets[] = {-1,0,1,-1,1,-1,0,1};

	int dest_x, dest_y;
	unsigned long long aggregate_flow = 0;
	// Check spatial neighbors
	for(int k=0;k<8;k++){
		dest_x = i + x_offsets[k] + 1;
		dest_y = j + y_offsets[k] + 1;
		aggregate_flow += F[dest_x][dest_y][7 - k];
	}

	// Run same condition as above for source, sink

	image_graph[i * width + j].excess += aggregate_flow;
}

__global__ void localRelabel(Pixel *image_graph, int height, int width)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int locali = i%BLOCK_SIZE, localj =j%BLOCK_SIZE;

	__shared__ int shared_heights[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
	__shared__ bool shared_flags[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

	shared_heights[locali + 1][localj + 1] = image_graph[i * width + j].height;
	shared_flags[locali + 1][localj + 1] = image_graph[i * width + j].is_active;

	__syncthreads();

	// Row major traversal of neighbors of a pixel (i,j)
	int x_offsets[] = {-1,-1,-1,0,0,1,1,1};
	int y_offsets[] = {-1,0,1,-1,1,-1,0,1};

	int dest_x, dest_y;
	unsigned long long aggregate_flow = 0;
	int min_height = INT_MAX;

	// Check spatial neighbors
	for(int i=0;i<8;i++){
		dest_x = locali + x_offsets[i] + 1;
		dest_y = localj + y_offsets[i] + 1;
		if( image_graph[dest_x * width + dest_y].is_active){
			min_height = min(min_height, shared_heights[dest_x][dest_y]);
		}
	}

	// Run same condition as above for source, sink

	image_graph[i * width + j].height = min_height;
}

__global__ void globalRelabel(Pixel *image_graph, int height, int width, int iteration)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int locali = i%BLOCK_SIZE, localj =j%BLOCK_SIZE;

	//No divergence
	if(iteration == 1){
		for (int l = 0; l < 8; l++)
			if(image_graph[i * width + j].neighbor_capacities[l] > image_graph[i * width + j].excess){
				image_graph[i * width + j].height = 1;
		}
	}
	else{
		__shared__ int shared_heights[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
		
		shared_heights[locali + 1][localj + 1] = image_graph[i * width + j].height;
		__syncthreads();

		bool satisfied = false;
		int dest_x, dest_y;

		// Row major traversal of neighbors of a pixel (i,j)
		int x_offsets[] = {-1,-1,-1,0,0,1,1,1};
		int y_offsets[] = {-1,0,1,-1,1,-1,0,1};

		for(int i1=0; i1<8; i1++){
			dest_x = locali + x_offsets[i1] + 1;
			dest_y = localj + y_offsets[i1] + 1;
			if(shared_heights[dest_x][dest_y] == iteration){
				satisfied = true;
				break;
			}
		}

		if(satisfied){
			shared_heights[locali + 1][localj + 1] = iteration + 1;
			image_graph[i * width + j].height = iteration + 1;
		}
	}
}

__global__ void initNeighbors(Pixel *imagegraph, int height, int width, unsigned long long* K){

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;

	int locali = i%BLOCK_SIZE, localj =j%BLOCK_SIZE;

	__shared__ Pixel block_pixels[BLOCK_SIZE + 2][BLOCK_SIZE + 2];
	block_pixels[locali + 1][localj + 1] = imagegraph[i * width + j];

	// Load pixels from boundary neighbors


	__syncthreads();

	// Row major traversal of neighbors of a pixel (i,j)
	int x_offsets[] = {-1,-1,-1,0,0,1,1,1};
	int y_offsets[] = {-1,0,1,-1,1,-1,0,1};

	unsigned long long max_k = 0;
	int dest_x, dest_y;

	for(int i=0; i<8; i++){
		dest_x = locali + x_offsets[i] + 1;
		dest_y = localj + y_offsets[i] + 1;
		block_pixels[locali + 1][localj + 1].neighbor_capacities[i] = B_function(block_pixels[locali + 1][localj + 1].pixel_value, block_pixels[dest_x][dest_y].pixel_value);
		max_k += block_pixels[locali + 1][localj + 1].neighbor_capacities[i];
	}
	max_k++;

	__shared__ unsigned long long blockmax;

	if(threadIdx.x == 0 && threadIdx.y == 0){
		blockmax = INT_MAX;
	}
	__syncthreads();

	atomicMax(&blockmax, max_k);
	__syncthreads();

	if(threadIdx.x == 0 && threadIdx.y == 0){
		atomicMax(K, blockmax);
	}

	// Won't work; copy neighbor wise
	imagegraph[i * width + j] = block_pixels[locali + 1][localj + 1];
}

//Also accept hard and soft constraints array
__global__ void initConstraints(Pixel *imagegraph, int height, int width, unsigned long long K){

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;

	// {p,S} edge
	imagegraph[i * width + j].neighbor_capacities[8] = (imagegraph[i* width + j].hard_constraint[i][j] == 0) * K
										+ (imagegraph[i* width + j].hard_constraint[i][j] == 1) * LAMBDA * R_function(imagegraph[i* width + j].pixel_value, -1);

	// {p,T} edge
	imagegraph[i* width + j].neighbor_capacities[9] = (imagegraph[i* width + j].hard_constraint[i][j] == -1) * K
										+ (imagegraph[i* width + j].hard_constraint[i][j] == 0) * LAMBDA * R_function(imagegraph[i* width + j].pixel_value, 1);
}


int main(int argc, char* argv[])
{
	int width, height;
	unsigned long long* K = new unsigned long long;
	*K = LLONG_MAX;
	bool* convergence_flag = new bool;

	unsigned char *image;
	Pixel *image_graph, *cuda_image_graph;
	
	ilInit();

	ILuint image_id = loadImage(argv[1], &image, width, height);
	int pixel_memsize = (width+1) * (height+1) * sizeof(Pixel);
	if(image_id == 0) {fprintf(stderr, "Error while reading image... aborting.\n"); exit(0);}

	//Pixel graph with padding to avoid convergence in kernels for boundary pixels
	image_graph = (Pixel*)malloc(pixel_memsize); 

	assert(hipSuccess == hipMalloc((void**) &cuda_image_graph, pixel_memsize));
	assert(hipSuccess == hipMemcpy(cuda_image_graph, image_graph, pixel_memsize, hipMemcpyHostToDevice));

	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks(width/BLOCK_SIZE, height/BLOCK_SIZE);
	
	// Load weights in graph using kernel call/host loops
	initNeighbors<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, height, width, K);
	initConstraints<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, height, width, *K);

	int iteration = 1;
	while(!convergence_flag){
		for(int i=0; i<HYPERk; i++){
			for(int j=0; j<HYPERm; j++){
				push<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, height, width);
				pull<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, width, height);
			}
			localRelabel<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, width, height);
		}
		globalRelabel<<<numBlocks, threadsPerBlock>>>(cuda_image_graph, width, height, iteration);
		iteration++;
	}

	// Load segmented image from graph using another kernel and display it

	return 0;	
}
